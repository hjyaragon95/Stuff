#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

extern "C"
{  
  __global__ void
  truncnorm_robert_kernel(
    
    float *x,      // Vector to contain returned samples 
    int n,         // Number of samples to retur
    float *mu,     // Vector of mu_s
    float *sigma,  // Vector of sigma_s
    float *a,      // Vector of lower-truncation values
    float *b,      // Vector of upper-truncation values
    int *mode,      // 1:(inf,inf); 2:(inf,b); 3:(a,inf);4:(a,b)
    int rng_a,     //input parameters as RNG seed constant
    int rng_b,     //input parameters as RNG seed constant
    int rng_c)      //input parameters as RNG seed constant
{
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) +threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;
    int nthreads <- prod(grid_dims)*prod(block_dims)
    double e = 4.71828183; double ai,bi,y,alpha,u,gx,z,t2; int accept;
    
    // if(n>=nthreads){idx=idx-nthreads}
    if(idx<n){
      if(mode[idx]==2){
        // set up the random number generating
        hiprandState rng;
	hiprand_init(rng_a+idx*rng_b,rng_c,0,&rng);
        x[idx]=mu[idx]+sqrt(sigma[idx])*hiprand_normal(&rng);
      }
      
      else if (mode[idx]==0){
        // set up the random number generating
        hiprandState rng;hiprand_init(rng_a+idx*rng_b,rng_c,0,&rng);
        //mirrow
         ai=(-b[idx]+mu[idx])/sqrt(sigma[idx]);
	 accept=0;
        //Then sample the truncated normal
        while(accept==0){
          //generate alpha
          alpha=(ai+sqrt(ai*ai+4.0))/2.0;
          //generate the z using inverse CDF
           y=hiprand_uniform(&rng); z=ai-1/alpha*log(1-y); t2=0;
          while(z<ai){
            t2=t2+1;hiprand_init(rng_a+idx*rng_b,rng_c+t2,0,&rng);
            y=hiprand_uniform(&rng);z=ai-1/alpha*log(1-y);  
          }
          //calculate g(x)
          if(ai<alpha){
	    gx=pow(e,-pow(alpha-z,2)/2);
          }
          else {
            gx=pow(e,-pow(ai-alpha,2)/2-pow(alpha-z,2)/2);
          }
          //decide whether to adopt z 
          u=hiprand_uniform(&rng);
          if(u<gx){
            accept=1;x[idx]=sqrt(sigma[idx])*z-mu[idx];x[idx]=-x[idx];
          }
        }
      }
      // mode==3:(a,inf)
      else if (mode[idx]==1){
        // set up the random number generating
        hiprandState rng;hiprand_init(rng_a+idx*rng_b,rng_c,0,&rng);
        //Then sample the truncated normal
        accept=0;ai=(a[idx]-mu[idx])/sqrt(sigma[idx]);
        while(accept==0){
          //generate alpha
         alpha=(ai+sqrt(ai*ai+4.0))/2.0;
          //generate the z using inverse CDF
          y=hiprand_uniform(&rng);z=ai-1/alpha*log(1-y);t2=0;
          while(z<ai){
            t2=t2+1;hiprand_init(rng_a+idx*rng_b,rng_c+t2,0,&rng);
            y=hiprand_uniform(&rng);z=ai-1/alpha*log(1-y);  
          }
          //calculate g(x)
          if(ai<alpha){
            gx=pow(e,(-pow((alpha-z),2)/2));
          }
          else {
            gx=pow(e,(pow(ai-alpha,2)/2))*pow(e,(-pow(alpha-z,2)/2));
          }
          //decide whether to adopt z 
          u=hiprand_uniform(&rng);
          if(u<gx){
            x[idx]=z;accept=1;x[idx]=sqrt(sigma[idx])*x[idx]+mu[idx];
          }
        } 
      }

      else if (mode[idx]==3){
        // set up the random number generating
        hiprandState rng;hiprand_init(rng_a+idx*rng_b,rng_c,0,&rng);
        ai=(a[idx]-mu[idx])/sqrt(sigma[idx]); bi=(b[idx]-mu[idx])/sqrt(sigma[idx]);
        //Then sample the truncated normal
        accept=0;
        while(accept==0){
          //generate the z using uniform
          z=ai+(bi-ai)*hiprand_uniform(&rng);
          //calculate g(x)
          if(ai<=0 && bi>=0){
            gx=pow(e,-z*z/2);
          }
          else if (bi<0) {
            gx=pow(e,((bi*bi-z*z)/2));
          }
          else if(ai>0){
            gx=pow(e,((ai*ai-z*z)/2));
          }
          //decide whether to adopt z 
          u=hiprand_uniform(&rng);
          if(u<gx){
            x[idx]=z;accept=1;x[idx]=sqrt(sigma[idx])*x[idx]+mu[idx];
            }
        } 
      }
    }  
    return;
  }  
  
} // END extern "C"




